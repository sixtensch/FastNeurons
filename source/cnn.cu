#include "hip/hip_runtime.h"
#include "cnn.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <cstdlib>
#include <cstdio>
#include <cstdint>
#include <ctime>

#include <random>
#include <chrono>





//~ Macro definitions

// Column major get
#define GET(data, x, y, height) (data[x * height + y])

#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define MAX(x, y) (((x) > (y)) ? (x) : (y))

#define PRINT_ERROR(format, ...)   printf("Error:   " format "\n", ##__VA_ARGS__);
#define PRINT_WARNING(format, ...) printf("Warning: " format "\n", ##__VA_ARGS__);
#define PRINT_INFO(format, ...)    printf("Info:    " format "\n", ##__VA_ARGS__);

#define CUDA_CHECK(stamp) { hipError_t _e = hipGetLastError(); if (_e != hipSuccess) { PRINT_ERROR("%i at [" stamp "]: %s", _e, hipGetErrorString(_e)); fflush(stdout); } }



//~ Math functions

__device__ float ReLU(num x)
{
	return x * (x > 0);
}

__device__ num ReLUDeriv(num x)
{
	return x > 0;
}

#define PRELU_A ((num)0.02)
__device__ num PReLU(num x)
{
	bool p = x > 0;
	return
		x * p +
		x * PRELU_A * !p;
}

__device__ num PReLUDeriv(num x)
{
	bool p = x > 0;
	return p + !p * PRELU_A;
}

num RandomNum(num min, num max)
{
	static std::mt19937 engine((unsigned int)time(0));
    
	std::uniform_real_distribution<num> distribution(min, max);
	return distribution(engine);
}



//~ Help functions

int ReadDataset(Dataset* dataset, int count, int width, int height, const char* pathImages, const char* pathLabels)
{
	FILE* fileImage = fopen(pathImages, "rb");
	FILE* fileLabel = fopen(pathLabels, "rb");
    
	if (fileImage == nullptr || fileLabel == nullptr)
	{
		return 1;
	}
    
	dataset->count = count;
	dataset->images = (Pixel*)malloc(count * width * height * sizeof(Pixel));
	dataset->labels = (Label*)malloc(count * sizeof(Label));
    
	fseek(fileImage, 16, SEEK_SET);
	fseek(fileLabel, 8, SEEK_SET);
    
	int r = (int)fread(dataset->images, count * width * height * sizeof(Pixel), 1, fileImage);
    r = (int)fread(dataset->labels, count * sizeof(Label), 1, fileLabel);
    
	fclose(fileImage);
	fclose(fileLabel);
    
	return 0;
}

// Provided with the example
void PrintResult(int confusion_matrix[10][10])
{
	// Print the confusion matrix
	printf("%15sPredicted label\n%10s", " ", " ");
	for (int col = 0; col < 10; col++)
		printf("%6d", col);
	printf("%10s\n", "Total");
	for (int n = 0; n < 70; n++)
		printf("%s", "-");
	printf("\nTrue label\n");
	int row_labels = 0;
	int total = 0;
	for (int row = 0; row < 10; row++) {
		row_labels = 0;
		printf("%10d", row);
		for (int col = 0; col < 10; col++) {
			printf("%6d", confusion_matrix[row][col]);
			row_labels += confusion_matrix[row][col];
		}
		printf("%10d\n", row_labels);
		total += row_labels;
	}
	for (int n = 0; n < 70; n++)
		printf("%s", "-");
	printf("\n%67s = %10d\n", "Total number of input images tested", total);
	for (int n = 0; n < 70; n++)
		printf("%s", "-");
	printf("\n");
}


void PrintImage(Pixel* pixels, int width, int sampling = 2, bool tendToMax = true, bool drawFrame = true)
{
	static const char characters[] = " .*#";
	static const byte characterCount = sizeof(characters) - 1;
    
	static char buffer[(32 * 2 + 3) * (32 + 2) + 1] = { '\0' };
	int i = 0;
    
#define BPRINT(c) buffer[i] = c; i++;
    
	if (drawFrame)
	{
		BPRINT('*');
		for (int f = 0; f < width / sampling; f++)
		{
			BPRINT('-');
			BPRINT('-');
		}
		BPRINT('*');
		BPRINT('\n');
	}
    
	for (int xp = 0; xp < width / sampling; xp++)
	{
		if (drawFrame)
		{
			BPRINT('|');
		}
        
		for (int yp = 0; yp < width / sampling; yp++)
		{
			unsigned int max = 0;
			unsigned int sum = 0;
            
			for (int xs = 0; xs < sampling; xs++)
			{
				for (int ys = 0; ys < sampling; ys++)
				{
					int x = xp * sampling + xs;
					int y = yp * sampling + xs;
                    
					sum += GET(pixels, x, y, width);
                    
					Pixel current = GET(pixels, x, y, width);
					if (current > max)
					{
						max = current;
					}
				}
			}
            
			Pixel value = 0;
			unsigned int average = sum / (sampling * sampling);
            
			if (tendToMax)
			{
				// Middle point between average and max, gives good legibility
				value = (Pixel)((average + max) / 2);
			}
			else
			{
				value = (Pixel)average;
			}
            
			BPRINT(characters[value / (256 / characterCount)]);
			BPRINT(characters[value / (256 / characterCount)]);
		}
        
		if (drawFrame)
		{
			BPRINT('|');
		}
        
		BPRINT('\n');
	}
    
	if (drawFrame)
	{
		BPRINT('*');
		for (int f = 0; f < width / sampling; f++)
		{
			BPRINT('-');
			BPRINT('-');
		}
		BPRINT('*');
		BPRINT('\n');
	}
    
	buffer[i] = '\0';
    
#undef BPRINT
    
	printf("%s", buffer);
}

void PrintImageFloating(num* pixels, int width, int sampling = 2, bool tendToMax = true, bool drawFrame = true)
{
	Pixel* buffer = (Pixel*)malloc(width * width * sizeof(Pixel));
    
	for (int x = 0; x < width; x++)
	{
		for (int y = 0; y < width; y++)
		{
			GET(buffer, x, y, width) = (Pixel)(MAX(MIN((int)(255 * GET(pixels, x, y, width)), 255), 0));
		}
	}
    
	PrintImage(buffer, width, sampling, tendToMax, drawFrame);
    
	free(buffer);
}



//~ Network help functions

int NetworkAddLayerHelper(Network* network, int networkSize, int stateSize)
{
	if (network->layerCount >= network->layerCapacity)
	{
		PRINT_ERROR("Maximum number of layers reached.\n");
		return -1;
	}
    
	int index = network->layerCount;
    
	network->networkSize += networkSize;
	network->stateSize += stateSize;
    
	network->layers[index].networkSize = networkSize;
	network->layers[index].stateSize = stateSize;
    
	network->layers[index].elementCount = 1;
	network->layers[index].elementWidth = 1;
	network->layers[index].elementHeight = 1;
    
	if (network->layerCount > 0)
	{
		network->layers[index].networkOffset =
			network->layers[index - 1].networkOffset +
			network->layers[index - 1].networkSize;
        
		network->layers[index].stateOffset =
			network->layers[index - 1].stateOffset +
			network->layers[index - 1].stateSize;
	}
	else
	{
		network->layers[index].networkOffset = 0;
		network->layers[index].stateOffset = 0;
	}
    
	network->layerCount++;
    
	return index;
}



//~ Network creation functions

void NetworkCreate(Network* network, int layerCapacity)
{
	network->layers = (Layer*)malloc(layerCapacity * sizeof(Layer));
	memset(network->layers, 0, layerCapacity * sizeof(Layer));
    
	network->networkSize = 0;
	network->stateSize = 0;
    
	network->layerCount = 0;
	network->layerCapacity = layerCapacity;
}

void NetworkDestroy(Network* network)
{
	free(network->layers);
}

void NetworkAddLayerInput2D(Network* network, int count, int width, int height, int padding)
{
	int networkSize = 0;
	int stateSize = count * (width + padding) * (height + padding);
    
	int index = NetworkAddLayerHelper(network, networkSize, stateSize);
	if (index < 0)
	{
		return;
	}
    
	network->layers[index].type = LayerTypeInput2D;
    
	network->layers[index].elementCount = count;
	network->layers[index].elementWidth = width + padding;
	network->layers[index].elementHeight = height + padding;
    
	network->layers[index].input2DPadding = padding;
}

void NetworkAddLayerConvolution2D(Network* network, int count, int kernelWidth, ActivationType activation)
{
	if (network->layerCount < 1 ||
		network->layers[network->layerCount - 1].elementWidth < kernelWidth ||
		network->layers[network->layerCount - 1].elementHeight < kernelWidth)
	{
		PRINT_ERROR("Convolution layer incompatible with previous layer.\n");
		return;
	}
    
	int oldCount = network->layers[network->layerCount - 1].elementCount;
	int width = network->layers[network->layerCount - 1].elementWidth - kernelWidth + 1;
	int height = network->layers[network->layerCount - 1].elementHeight - kernelWidth + 1;
    
	int weightsSize = oldCount * count * kernelWidth * kernelWidth;
    
	int networkSize = weightsSize + count; // Weigths + biases
	int stateSize = count * width * height;
    
	int index = NetworkAddLayerHelper(network, networkSize, stateSize);
	if (index < 0)
	{
		return;
	}
    
	network->layers[index].type = LayerTypeConvolution2D;
    
	network->layers[index].elementCount = count;
	network->layers[index].elementWidth = width;
	network->layers[index].elementHeight = height;
    
	network->layers[index].conv2DActivationType = activation;
	network->layers[index].conv2DKernelWidth = kernelWidth;
	network->layers[index].conv2DBiasOffset = weightsSize;
}

void NetworkAddLayerPooling2D(Network* network, int eyeWidth, PoolingType pooling)
{
	if (network->layerCount < 1 ||
		network->layers[network->layerCount - 1].elementWidth < eyeWidth ||
		network->layers[network->layerCount - 1].elementHeight < eyeWidth)
	{
		PRINT_ERROR("Pooling layer incompatible with previous layer.\n");
		return;
	}
    
	if (network->layers[network->layerCount - 1].elementWidth % eyeWidth > 0 ||
		network->layers[network->layerCount - 1].elementHeight % eyeWidth > 0)
	{
		PRINT_WARNING("Pooling layer will delete information.\n");
	}
    
	int count = network->layers[network->layerCount - 1].elementCount;
	int width = network->layers[network->layerCount - 1].elementWidth / eyeWidth;
	int height = network->layers[network->layerCount - 1].elementHeight / eyeWidth;
    
	int networkSize = 0;
	int stateSize = count * width * height;
    
	int index = NetworkAddLayerHelper(network, networkSize, stateSize);
	if (index < 0)
	{
		return;
	}
    
	network->layers[index].type = LayerTypePooling2D;
    
	network->layers[index].elementCount = count;
	network->layers[index].elementWidth = width;
	network->layers[index].elementHeight = height;
    
	network->layers[index].pool2DEyeWidth = eyeWidth;
	network->layers[index].pool2DType = pooling;
}

void NetworkAddLayerFullyConnected(Network* network, int count, ActivationType activation)
{
	if (network->layerCount < 1)
	{
		PRINT_ERROR("Fully connected layer cannot be first layer.\n");
		return;
	}
    
	int oldCountTotal =
		network->layers[network->layerCount - 1].elementCount *
		network->layers[network->layerCount - 1].elementWidth *
		network->layers[network->layerCount - 1].elementHeight;
    
	int weightsSize = oldCountTotal * count;
    
	int networkSize = weightsSize + count; // Weights + biases
	int stateSize = count;
    
	int index = NetworkAddLayerHelper(network, networkSize, stateSize);
	if (index < 0)
	{
		return;
	}
    
	network->layers[index].type = LayerTypeFullyConnected;
    
	network->layers[index].elementCount = count;
	network->layers[index].elementWidth = 1;
	network->layers[index].elementHeight = 1;
    
	network->layers[index].fcActivationType = activation;
	network->layers[index].fcBiasOffset = weightsSize;
}

void NetworkInstanceCreate(Network* network, NetworkInstance* instance, bool random = true)
{
	hipMalloc(&instance->deviceMemory, network->networkSize * sizeof(num));
    
	CUDA_CHECK("NetworkInstanceCreate Allocation");
    
	if (!random)
	{
		return;
	}
    
	num* stage = (num*)malloc(network->networkSize * sizeof(num));
	memset(stage, 0, network->networkSize * sizeof(num));
    
	Layer* layer = nullptr;
	Layer* previous = nullptr;
    
	for (int i = 0; i < network->layerCount; i++)
	{
		layer = &network->layers[i];
        
		switch (layer->type)
		{
            case LayerTypeConvolution2D:
            {
                num factor = (num)sqrt(2.0 / (layer->conv2DKernelWidth *
                                              layer->conv2DKernelWidth *
                                              (previous->elementCount + layer->elementCount)));
                
                for (int j = 0; j < layer->conv2DBiasOffset; j++)
                {
                    stage[layer->networkOffset + j] = factor * RandomNum(-1, 1);
                }
            }
            break;
            
            case LayerTypeFullyConnected:
            {
                num factor = (num)sqrt(2.0 / (previous->elementCount *
                                              previous->elementWidth *
                                              previous->elementHeight +
                                              layer->elementCount));
                
                for (int j = 0; j < layer->fcBiasOffset; j++)
                {
                    stage[layer->networkOffset + j] = factor * RandomNum(-1, 1);
                }
            }
            break;
		}
        
		previous = layer;
	}
    
	hipMemcpy(instance->deviceMemory, stage, network->networkSize * sizeof(num), hipMemcpyHostToDevice);
    
	CUDA_CHECK("NetworkInstanceCreate Memcpy");
    
	free(stage);
}

void NetworkInstanceDestroy(NetworkInstance* instance)
{
	hipFree(instance->deviceMemory);
	instance->deviceMemory = nullptr;
    
	CUDA_CHECK("NetworkInstanceDestroy");
}

void NetworkStateCreate(Network* network, NetworkState* state, int count)
{
	state->count = count;
	hipMalloc(&state->deviceMemory, count * network->stateSize * sizeof(num));
    
	CUDA_CHECK("NetworkStateCreate");
}

void NetworkStateDestroy(NetworkState* state)
{
	hipFree(state->deviceMemory);
	state->count = 0;
	state->deviceMemory = nullptr;
    
	CUDA_CHECK("NetworkStateDestroy");
}



//- Kernels

//~ Forward propagation kernels

template<Activation Activation>
__global__ void Convolve2DForward(num* weights, num* biases, num* input, num* output, int stateSize,
                                  int inWidth, int inHeight, int inCount,
                                  int outWidth, int outHeight, int outCount,
                                  int kernelWidth)
{
	num accumulator = 0;
    
	int kernelSize = kernelWidth * kernelWidth;
	int inputSize = inWidth * inHeight;
	int outputSize = outWidth * outHeight;
    
	// Indexing
	int batchIndex = blockIdx.y;
	int outIndex = blockIdx.x;
	int outX = threadIdx.x;
	int outY = threadIdx.y;
    
	for (int i = 0; i < inCount; i++)
	{
		for (int x = 0; x < kernelWidth; x++)
		{
			for (int y = 0; y < kernelWidth; y++)
			{
				// Indexing
				int inX = threadIdx.x + x;
				int inY = threadIdx.y + y;
				int kernelX = x;
				int kernelY = y;
				int inIndex = i;
                
				// Kernels are serialized by outIndex, where all the weights at [0, 0] are adjacent
				num weight = weights[
                                     kernelX * kernelWidth + kernelY +
                                     inIndex * kernelSize +
                                     outIndex * kernelSize * inCount];
                
				// Images are serialized column-major in series
				accumulator += weight * input[
                                              inX * inHeight +
                                              inY +
                                              inIndex * inputSize +
                                              batchIndex * stateSize];
			}
		}
	}
    
	num result = Activation(accumulator + biases[blockIdx.x]);
    
	output[
           outX * outHeight +
           outY +
           outIndex * outputSize +
           batchIndex * stateSize] = result;
}

__global__ void Pool2DForwardMax(num* input, num* output, int stateSize, int count,
                                 int inWidth, int inHeight, int outWidth, int outHeight, int eyeWidth)
{
	num max = 0;
    
	int inputSize = inWidth * inHeight;
	int outputSize = outWidth * outHeight;
    
	int inputOffset = inputSize * blockIdx.x + stateSize * blockIdx.y;
    
	for (int x = 0; x < eyeWidth; x++)
	{
		for (int y = 0; y < eyeWidth; y++)
		{
			num value = input[(threadIdx.x * eyeWidth + x) * inHeight +
                              (threadIdx.y * eyeWidth + y) +
                              inputOffset];
            
			bool greater = value > max;
			max = greater * value + !greater * max;
		}
	}
    
	output[threadIdx.x * outHeight +
           threadIdx.y +
           outputSize * blockIdx.x +
           stateSize * blockIdx.y] = max;
}

template<Activation Activation>
__global__ void FullyConnectForward(num* weights, num* biases, num* input, num* output, int stateSize,
                                    int inCount, int outCount)
{
	num accumulator = 0;
    
	for (int i = 0; i < inCount; i++)
	{
		accumulator += input[i + stateSize * blockIdx.x] * weights[threadIdx.x + i * blockDim.x];
	}
    
	num result = Activation(accumulator + biases[threadIdx.x]);
    
	output[threadIdx.x + stateSize * blockIdx.x] = result;
}

//~ Backpropagation kernels

// Initialization for output layers
// Computes error based on a type of desired value (cost)

__global__ void SoftmaxComputeErrors(num* output, num* errors, Label* labels, int stateSize, int count)
{
	__shared__ num inner;
    
	int label = (int)labels[blockIdx.x];
    
	num sum = 0;
	num myValue = output[threadIdx.x + blockIdx.x * stateSize];
	for (int i = 0; i < count; i++)
	{
		sum += expf(output[i + blockIdx.x * stateSize] - myValue);
	}
    
	num error = (num)1.0 / sum;
    
	if (label == threadIdx.x)
	{
		inner = error;
	}
    
	__syncthreads();
    
	atomicAdd(&inner, -(error * error));
    
	__syncthreads();
    
	errors[threadIdx.x + blockIdx.x * stateSize] += error * ((threadIdx.x == label) - error - inner);
}

// Layer backpropagation
// Calculates error for input layer based on error of output layer, as well as connecting deltas for weights and biases

template<Activation ActivationDeriv>
__global__ void Convolve2DBackwardErrors(num* weights, num* input, num* inErrors, num* outErrors, int stateSize,
                                         int inWidth, int inHeight, int inCount,
                                         int outWidth, int outHeight, int outCount,
                                         int kernelWidth)
{
	int kernelSize = kernelWidth * kernelWidth;
	int inputSize = inWidth * inHeight;
	int outputSize = outWidth * outHeight;
    
	int batchIndex = blockIdx.y;
	int inImage = blockIdx.x;
	int inX = threadIdx.x;
	int inY = threadIdx.y;
    
	num accumulator = 0.0f;
    
	for (int i = 0; i < outCount; i++)
	{
		for (int x = 0; x < kernelWidth; x++)
		{
			for (int y = 0; y < kernelWidth; y++)
			{
				int outX = -x + threadIdx.x;
				int outY = -y + threadIdx.y;
				int kernelX = x;
				int kernelY = y;
				int outImage = i;
                
				bool inside =
					outX >= 0 && outX < outWidth &&
					outY >= 0 && outY < outHeight;
                
				num weight = weights[
                                     kernelX * kernelWidth + kernelY +
                                     inImage * kernelSize +
                                     outImage * kernelSize * inCount];
                
				int outIndex = 
					outX * outHeight +
					outY +
					outImage * outputSize +
					batchIndex * stateSize;
                
				accumulator += inside * weight * outErrors[inside * outIndex];
			}
		}
	}
    
	int inIndex = inX * inHeight +
		inY +
		inImage * inputSize +
		batchIndex * stateSize;
    
	inErrors[inIndex] = accumulator * ActivationDeriv(input[inIndex]);
}

__global__ void Convolve2DBackwardBiases(num* outErrors, num* biasDeltas, int stateSize, int networkSize, int outWidth, int outHeight)
{
	num sum = 0;
    
	for (int x = 0; x < outWidth; x++)
	{
		for (int y = 0; y < outHeight; y++)
		{
			sum += outErrors[
                             x * outWidth + y +
                             threadIdx.x * outWidth * outHeight + 
                             blockIdx.x * stateSize];
		}
	}
    
	biasDeltas[threadIdx.x + blockIdx.x * networkSize] += sum;
}

__global__ void Convolve2DBackwardWeights(num* input, num* outErrors, num* weightDeltas, int stateSize, int networkSize,
                                          int inWidth, int inHeight, int inCount,
                                          int outWidth, int outHeight, int outCount,
                                          int kernelWidth)
{
	num accumulator = 0;
    
	int kernelSize = kernelWidth * kernelWidth;
	int inputSize = inWidth * inHeight;
	int outputSize = outWidth * outHeight;
    
	// Indexing
	int batchIndex = blockIdx.z;
	int inIndex = blockIdx.x;
	int outIndex = blockIdx.y;
	int kernelX = threadIdx.x;
	int kernelY = threadIdx.y;
    
	for (int x = 0; x < outWidth; x++)
	{
		for (int y = 0; y < outHeight; y++)
		{
			// Indexing
			int inX = threadIdx.x + x;
			int inY = threadIdx.y + y;
			int outX = x;
			int outY = y;
            
			// Kernels are serialized by outIndex, where all the outErrors at [0, 0] are adjacent
			num weight = outErrors[
                                   outX * outHeight + 
                                   outY +
                                   outIndex * outputSize +
                                   batchIndex * stateSize];
            
			// Images are serialized column-major in series
			accumulator += weight * input[
                                          inX * inHeight +
                                          inY +
                                          inIndex * inputSize +
                                          batchIndex * stateSize];
		}
	}
    
	weightDeltas[
                 kernelX * kernelWidth + kernelY +
                 inIndex * kernelSize +
                 outIndex * kernelSize * inCount +
                 batchIndex * networkSize] += accumulator;
}

__global__ void Pool2DBackwardMax(num* input, num* inErrors, num* outErrors, int stateSize, int count,
                                  int inWidth, int inHeight, int outWidth, int outHeight, int eyeWidth)
{
	int xMax = 0;
	int yMax = 0;
    
	int inSize = inWidth * inHeight;
	int outSize = outWidth * outHeight;
    
	for (int x = 0; x < eyeWidth; x++)
	{
		for (int y = 0; y < eyeWidth; y++)
		{
			num current = input[(threadIdx.x * eyeWidth + x) * inHeight +
                                (threadIdx.y * eyeWidth + y) +
                                blockIdx.x * inSize +
                                blockIdx.y * stateSize];
            
			num comp = input[(threadIdx.x * eyeWidth + xMax) * inHeight +
                             (threadIdx.y * eyeWidth + yMax) +
                             blockIdx.x * inSize +
                             blockIdx.y * stateSize];
            
			bool greater = current > comp;
            
			xMax += greater * (x - xMax);
			yMax += greater * (y - yMax);
		}
	}
    
	num error = outErrors[
                          threadIdx.x * outHeight +
                          threadIdx.y +
                          blockIdx.x * outSize +
                          blockIdx.y * stateSize];
    
	inErrors[(threadIdx.x * eyeWidth + xMax) * inHeight +
             (threadIdx.y * eyeWidth + yMax) +
             blockIdx.x * inSize +
             blockIdx.y * stateSize] = error;
    
}

template<Activation Activation>
__global__ void FullyConnectBackwardErrors(num* weights, num* input, num* inErrors, num* outErrors, int stateSize,
                                           int inCount, int outCount)
{
	num accumulator = 0;
    
	for (int i = 0; i < outCount; i++)
	{
		accumulator += outErrors[i + blockIdx.x * stateSize] * weights[i + threadIdx.x * outCount];
	}
    
	inErrors[threadIdx.x + blockIdx.x * stateSize] += accumulator * Activation(input[threadIdx.x + blockIdx.x * stateSize]);
}

__global__ void FullyConnectBackwardBiases(num* outErrors, num* biasDeltas, int stateSize, int networkSize, int count)
{
	biasDeltas[threadIdx.x + blockIdx.x * networkSize] += outErrors[threadIdx.x + blockIdx.x * stateSize];
}

__global__ void FullyConnectBackwardWeights(num* input, num* outErrors, num* weightDeltas, int stateSize, int networkSize,
                                            int inSize, int outSize)
{
	num a = input[threadIdx.x +
                  blockIdx.y * stateSize];
    
	num b = outErrors[blockIdx.x +
                      blockIdx.y * stateSize];
    
	weightDeltas[blockIdx.x +
                 threadIdx.x * gridDim.x +
                 blockIdx.y * networkSize] += a * b;
}



//~ Misc kernels

__global__ void ApplyDelta(num* network, num* deltas, num coefficient, int networkSize, int batchSize)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
    
	if (id > networkSize)
	{
		return;
	}
    
	num delta = 0;
    
	for (int i = 0; i < batchSize; i++)
	{
		delta += deltas[id + networkSize * i];
	}
    
	deltas[id] = delta;
	network[id] += delta * coefficient;
}



//~ Network activation functions

void NetworkForwardPropagation(Network* network, NetworkInstance* instance, NetworkState* state, Dataset* dataset, int datasetOffset, int batchSize)
{
	static num buffer[128 * 128];
    
	Layer* layer = nullptr;
	Layer* previous = nullptr;
    
	for (int i = 0; i < network->layerCount; i++)
	{
		layer = &network->layers[i];
        
		switch (layer->type)
		{
            case LayerTypeInput2D:
            {
                int width = layer->elementWidth - layer->input2DPadding;
                int height = layer->elementHeight - layer->input2DPadding;
                
                // Reset the buffer, filling in any eventual padding
                int inSize = width * height;
                int outSize = layer->elementWidth * layer->elementHeight;
                memset(buffer, 0, outSize * sizeof(Pixel));
                
                int padStart = layer->input2DPadding / 2;
                
                for (int j = 0; j < batchSize; j++)
                {
                    Pixel* in = dataset->images + (datasetOffset + j) * width * height;
                    
                    num mean = 0;
                    num std = 0;
                    
                    for (int x = 0; x < width; x++)
                    {
                        for (int y = 0; y < height; y++)
                        {
                            Pixel sample = in[x * height + y];
                            mean += sample;
                            std += sample * sample;
                        }
                    }
                    
                    mean /= inSize;
                    std = (num)sqrt(std / inSize - mean * mean);
                    
                    num stdInv = 1.0f / std;
                    
                    for (int x = 0; x < width; x++)
                    {
                        for (int y = 0; y < height; y++)
                        {
                            buffer[(x + padStart) * (height + layer->input2DPadding) + padStart + y] = (num)(in[x * height + y] - mean) * stdInv;
                        }
                    }
                    
                    hipMemcpy(state->deviceMemory + j * network->stateSize + layer->stateOffset, buffer, outSize * sizeof(num), hipMemcpyHostToDevice);
                }
                
                CUDA_CHECK("Forward Input2D");
            }
            break;
            
            case LayerTypeConvolution2D:
            {
                dim3 dimBlock;
                dimBlock.x = layer->elementWidth;
                dimBlock.y = layer->elementHeight;
                dimBlock.z = 1;
                
                dim3 dimGrid;
                dimGrid.x = layer->elementCount;
                dimGrid.y = batchSize;
                dimGrid.z = 1;
                
                Convolve2DForward<ReLU>
                    <<<dimGrid, dimBlock>>> (instance->deviceMemory + layer->networkOffset,
                                             instance->deviceMemory + layer->networkOffset + layer->conv2DBiasOffset,
                                             state->deviceMemory + previous->stateOffset,
                                             state->deviceMemory + layer->stateOffset,
                                             network->stateSize,
                                             previous->elementWidth, previous->elementHeight, previous->elementCount,
                                             layer->elementWidth, layer->elementHeight, layer->elementCount,
                                             layer->conv2DKernelWidth);
                
                CUDA_CHECK("Forward Convolution2D");
            }
            break;
            
            case LayerTypePooling2D:
            {
                dim3 dimBlock;
                dimBlock.x = layer->elementWidth;
                dimBlock.y = layer->elementHeight;
                dimBlock.z = 1;
                
                dim3 dimGrid;
                dimGrid.x = layer->elementCount;
                dimGrid.y = batchSize;
                dimGrid.z = 1;
                
                Pool2DForwardMax<<<dimGrid, dimBlock>>>(state->deviceMemory + previous->stateOffset,
                                                        state->deviceMemory + layer->stateOffset,
                                                        network->stateSize,
                                                        layer->elementCount,
                                                        previous->elementWidth, previous->elementHeight,
                                                        layer->elementWidth, layer->elementHeight,
                                                        layer->pool2DEyeWidth);
                
                CUDA_CHECK("Forward Pooling2D");
            }
            break;
            
            case LayerTypeFullyConnected:
            {
                dim3 dimBlock;
                dimBlock.x = layer->elementCount;
                dimBlock.y = 1;
                dimBlock.z = 1;
                
                dim3 dimGrid;
                dimGrid.x = batchSize;
                dimGrid.y = 1;
                dimGrid.z = 1;
                
                FullyConnectForward<ReLU>
                    <<<dimGrid, dimBlock>>>(instance->deviceMemory + layer->networkOffset,
                                            instance->deviceMemory + layer->networkOffset + layer->fcBiasOffset,
                                            state->deviceMemory + previous->stateOffset,
                                            state->deviceMemory + layer->stateOffset,
                                            network->stateSize,
                                            previous->elementWidth * previous->elementHeight * previous->elementCount,
                                            layer->elementCount);
                
                CUDA_CHECK("Forward FC");
            }
            break;
		}
        
#if DEBUG_PRINT
		if (i == 0)
		{
			printf("\n\n");
			hipMemcpy(buffer, state->deviceMemory + layer->stateOffset, layer->elementWidth * layer->elementHeight * sizeof(num), hipMemcpyDeviceToHost);
			PrintImageFloating(buffer, layer->elementWidth, 2);
		}
        
		if (i == 2)
		{
			for (int j = 0; j < 3; j++)
			{
				hipMemcpy(buffer, state->deviceMemory + layer->stateOffset + layer->elementWidth * layer->elementHeight * j, layer->elementWidth * layer->elementHeight * sizeof(num), hipMemcpyDeviceToHost);
				PrintImageFloating(buffer, layer->elementWidth, 1);
			}
		}
#endif
        
		previous = layer;
	}
}

void NetworkLoadOutputSoftmax(Network* network, NetworkState* state, NetworkState* errors, Label* correct, int batchSize, Label* labelBuffer)
{
	//- Validation
    
	bool freeLabelBuffer = false;
    
	if (labelBuffer == nullptr)
	{
		freeLabelBuffer = true;
		hipMalloc(&labelBuffer, batchSize * sizeof(Label));
		CUDA_CHECK("Load output softmax allocation");
	}
    
	hipMemcpy(labelBuffer, correct, batchSize * sizeof(Label), hipMemcpyHostToDevice);
	CUDA_CHECK("Load output softmax copy");
    
	// Seed the error state with softmax errors in the output layer corresponding with the correct label
	Layer* outputLayer = &network->layers[network->layerCount - 1];
    
	SoftmaxComputeErrors<<<batchSize, outputLayer->elementCount>>>(state->deviceMemory + outputLayer->stateOffset,
                                                                   errors->deviceMemory + outputLayer->stateOffset,
                                                                   labelBuffer,
                                                                   network->stateSize,
                                                                   outputLayer->elementCount);
    
	CUDA_CHECK("Load output softmax");
    
	if (freeLabelBuffer)
	{
		hipFree(labelBuffer);
		CUDA_CHECK("Load output softmax deallocation");
	}
}

void NetworkBackwardPropagation(Network* network, NetworkInstance* instance, NetworkState* state, NetworkState* errors, num* deltas, int batchSize)
{
	// Clear the delta array
    
	hipMemset(deltas, 0, network->networkSize * batchSize * sizeof(num));
	CUDA_CHECK("Train deltas memset");
    
    
    
	//- Propagation
    
	Layer* layer = nullptr;
	Layer* previous = nullptr;
    
	for (int i = network->layerCount - 1; i >= 1; i--)
	{
		layer = &network->layers[i];
		previous = &network->layers[i - 1];
        
		bool backpropagateErrors = i > 1;
        
		switch (layer->type)
		{
            case LayerTypeInput2D:
            {
                // Shouldn't be hit
                PRINT_ERROR("Hit input layer in backpropagation.");
            }
            break;
            
            case LayerTypeConvolution2D:
            {
                dim3 dimBlock;
                dim3 dimGrid;
                
                if (backpropagateErrors)
                {
                    dimBlock.x = previous->elementWidth;
                    dimBlock.y = previous->elementHeight;
                    dimBlock.z = 1;
                    
                    dimGrid.x = previous->elementCount;
                    dimGrid.y = batchSize;
                    dimGrid.z = 1;
                    
                    Convolve2DBackwardErrors<ReLUDeriv>
                        << <dimGrid, dimBlock >> > (instance->deviceMemory + layer->networkOffset,
                                                    state->deviceMemory + previous->stateOffset,
                                                    errors->deviceMemory + previous->stateOffset,
                                                    errors->deviceMemory + layer->stateOffset,
                                                    network->stateSize,
                                                    previous->elementWidth, previous->elementHeight, previous->elementCount,
                                                    layer->elementWidth, layer->elementHeight, layer->elementCount,
                                                    layer->conv2DKernelWidth);
                    
                    CUDA_CHECK("Backward Convolution2D errors");
                }
                
                dimBlock.x = layer->elementCount;
                dimBlock.y = 1;
                dimBlock.z = 1;
                
                dimGrid.x = batchSize;
                dimGrid.y = 1;
                dimGrid.z = 1;
                
                Convolve2DBackwardBiases
                    <<<dimGrid, dimBlock>>>(errors->deviceMemory + layer->stateOffset,
                                            deltas + layer->networkOffset + layer->conv2DBiasOffset,
                                            network->stateSize, network->networkSize,
                                            layer->elementWidth, layer->elementHeight);
                
                CUDA_CHECK("Backward Convolution2D biases");
                
                dimBlock.x = layer->conv2DKernelWidth;
                dimBlock.y = layer->conv2DKernelWidth;
                dimBlock.z = 1;
                
                dimGrid.x = previous->elementCount;
                dimGrid.y = layer->elementCount;
                dimGrid.z = batchSize;
                
                Convolve2DBackwardWeights
                    <<<dimGrid, dimBlock>>>(state->deviceMemory + previous->stateOffset,
                                            errors->deviceMemory + layer->stateOffset,
                                            deltas + layer->networkOffset,
                                            network->stateSize, network->networkSize,
                                            previous->elementWidth, previous->elementHeight, previous->elementCount,
                                            layer->elementWidth, layer->elementHeight, layer->elementCount,
                                            layer->conv2DKernelWidth);
                
                CUDA_CHECK("Backward Convolution2D weights");
            }
            break;
            
            case LayerTypePooling2D:
            {
                dim3 dimBlock;
                dim3 dimGrid;
                
                if (backpropagateErrors)
                {
                    dimBlock.x = layer->elementWidth;
                    dimBlock.y = layer->elementHeight;
                    dimBlock.z = 1;
                    
                    dimGrid.x = layer->elementCount;
                    dimGrid.y = batchSize;
                    dimGrid.z = 1;
                    
                    Pool2DBackwardMax
                        << <dimGrid, dimBlock >> > (state->deviceMemory + previous->stateOffset,
                                                    errors->deviceMemory + previous->stateOffset,
                                                    errors->deviceMemory + layer->stateOffset,
                                                    network->stateSize,
                                                    layer->elementCount,
                                                    previous->elementWidth, previous->elementHeight,
                                                    layer->elementWidth, layer->elementHeight,
                                                    layer->pool2DEyeWidth);
                    
                    CUDA_CHECK("Backward Pooling2D errors");
                }
            }
            break;
            
            case LayerTypeFullyConnected:
            {
                dim3 dimBlock;
                dim3 dimGrid;
                
                if (backpropagateErrors)
                {
                    dimBlock.x = previous->elementCount * previous->elementWidth * previous->elementHeight;
                    dimBlock.y = 1;
                    dimBlock.z = 1;
                    
                    dimGrid.x = batchSize;
                    dimGrid.y = 1;
                    dimGrid.z = 1;
                    
                    FullyConnectBackwardErrors<ReLUDeriv>
                        <<<dimGrid, dimBlock>>>(instance->deviceMemory + layer->networkOffset,
                                                state->deviceMemory + previous->stateOffset,
                                                errors->deviceMemory + previous->stateOffset,
                                                errors->deviceMemory + layer->stateOffset,
                                                network->stateSize,
                                                previous->elementWidth * previous->elementHeight * previous->elementCount,
                                                layer->elementCount);
                    
                    CUDA_CHECK("Backward FC errors");
                }
                
                dimBlock.x = layer->elementCount;
                dimBlock.y = 1;
                dimBlock.z = 1;
                
                dimGrid.x = batchSize;
                dimGrid.y = 1;
                dimGrid.z = 1;
                
                FullyConnectBackwardBiases
                    <<<dimGrid, dimBlock>>>(errors->deviceMemory + layer->stateOffset,
                                            deltas + layer->networkOffset + layer->fcBiasOffset,
                                            network->stateSize, network->networkSize,
                                            layer->elementCount);
                
                CUDA_CHECK("Backward FC biases");
                
                dimBlock.x = previous->elementWidth * previous->elementHeight * previous->elementCount;
                dimBlock.y = 1;
                dimBlock.z = 1;
                
                dimGrid.x = layer->elementCount;
                dimGrid.y = batchSize;
                dimGrid.z = 1;
                
                FullyConnectBackwardWeights
                    <<<dimGrid, dimBlock>>>(state->deviceMemory + previous->stateOffset,
                                            errors->deviceMemory + layer->stateOffset,
                                            deltas + layer->networkOffset,
                                            network->stateSize, network->networkSize,
                                            previous->elementWidth * previous->elementHeight * previous->elementCount,
                                            layer->elementCount);
                
                CUDA_CHECK("Backward FC weights");
            }
            break;
            
		}
	}
}

void NetworkApplyDeltas(Network* network, NetworkInstance* instance, num* deltas, num alpha, int batchSize)
{
	//- Applying deltas
    
	int applyBlockSize = 512;
	int applyBlockCount = (network->networkSize - 1) / 512 + 1;
    
	hipDeviceSynchronize();
    
	ApplyDelta<<<applyBlockCount, applyBlockSize>>>(
                                                    instance->deviceMemory,
                                                    deltas,
                                                    alpha / batchSize,
                                                    network->networkSize,
                                                    batchSize);
    
	CUDA_CHECK("Apply weights");
    
	hipDeviceSynchronize();
}



//~ Training functions

void NetworkTrain(Network* network, NetworkInstance* instance, Dataset* dataset, int desiredBatchSize, num alpha)
{
	// Create a state capable of storing all simultaneous network features for one batch
	NetworkState state;
	NetworkStateCreate(network, &state, desiredBatchSize);
    
	// Create a corresponding state used for calculation of error
	NetworkState errors;
	NetworkStateCreate(network, &errors, desiredBatchSize);
    
	// Create a delta vector for the network
	num* deltas;
	hipMalloc(&deltas, network->networkSize * desiredBatchSize * sizeof(num));
	CUDA_CHECK("Train deltas allocation");
    
	Label* correctLabels;
	hipMalloc(&correctLabels, desiredBatchSize * sizeof(Label));
	CUDA_CHECK("Train label allocation");
    
	// The number of batches
	// If it does not add up evenly, the last batch will be smaller
	int batches = (dataset->count - 1) / desiredBatchSize + 1;
    
	// Working buffer used for padding inputs
	num* buffer;
	hipHostMalloc(&buffer, network->layers[0].stateSize * sizeof(num));
	CUDA_CHECK("Train buffer allocation");
    
	// Main training loop
	for (int batch = 0; batch < batches; batch++)
	{
		// Top level parameters for this batch
		int datasetOffset = batch * desiredBatchSize;
		int batchSize = MIN(desiredBatchSize, dataset->count - datasetOffset);
        
		// Clear the state and error arrays
		hipMemset(state.deviceMemory, 0, network->stateSize * batchSize * sizeof(num));
		hipMemset(errors.deviceMemory, 0, network->stateSize * batchSize * sizeof(num));
		CUDA_CHECK("Train states memset");
        
        
        
		//- Forward propagation
        
		NetworkForwardPropagation(network, instance, &state, dataset, datasetOffset, batchSize);
        
        
        
		//- Validation
        
		NetworkLoadOutputSoftmax(network, &state, &errors, dataset->labels + datasetOffset, batchSize, correctLabels);
        
        
        
		//- Backpropagation
        
		NetworkBackwardPropagation(network, instance, &state, &errors, deltas, batchSize);
        
        
        
		//- Apply deltas
        
		NetworkApplyDeltas(network, instance, deltas, alpha, batchSize);
	}
    
	// Deinitialize
	hipHostFree(buffer);
	hipFree(deltas);
	hipFree(correctLabels);
	CUDA_CHECK("Train free");
    
	NetworkStateDestroy(&state);
	NetworkStateDestroy(&errors);
}



//~ Testing

void NetworkTest(Network* network, NetworkInstance* instance, Dataset* dataset, bool printMatrix)
{
	int desiredBatchSize = 300;
    
	NetworkState state;
	NetworkStateCreate(network, &state, desiredBatchSize);
    
	// The number of batches
	// If it does not add up evenly, the last batch will be smaller
	int batches = (dataset->count - 1) / desiredBatchSize + 1;
    
	int confusionMatrix[10][10] = { 0 };
    
	int correct = 0;
	int total = dataset->count;
    
	for (int batch = 0; batch < batches; batch++)
	{
		// Top level parameters for this batch
		int datasetOffset = batch * desiredBatchSize;
		int batchSize = MIN(desiredBatchSize, dataset->count - datasetOffset);
        
		// Clear the state and error arrays
		hipMemset(state.deviceMemory, 0, network->stateSize * batchSize * sizeof(num));
		CUDA_CHECK("Train states memset");
        
		NetworkForwardPropagation(network, instance, &state, dataset, datasetOffset, batchSize);
        
		// Seed the error state with softmax errors in the output layer corresponding with the correct label
		Layer* outputLayer = &network->layers[network->layerCount - 1];
        
		for (int i = 0; i < batchSize; i++)
		{
			num outs[10];
			hipMemcpy(outs, state.deviceMemory + outputLayer->stateOffset + network->stateSize * i, 10 * sizeof(num), hipMemcpyDeviceToHost);
            
			int max = 0;
			for (int j = 1; j < 10; j++)
			{
				if (outs[j] > outs[max])
				{
					max = j;
				}
			}
            
			int label = (int)dataset->labels[batch * desiredBatchSize + i];
            
			correct += label == max;
            
			confusionMatrix[label][max] += 1;
		}
	}
    
	if (printMatrix)
	{
		PrintResult(confusionMatrix);
		printf("\n");
	}
    
	printf("Correct guesses: %i / %i (%.4f%%)\n", correct, total, (double)correct / total * 100);
    
	NetworkStateDestroy(&state);
}
